#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE 1024
#define LOW 12
#define HIGH 30

__global__ void cu_examine(double*, int, int*);

extern "C" int checkInRange(double *coords, int num)  
{
	int answer;
	int memory = num*sizeof(double);
	double *d_coords;
	int * counter;
	int error = 0;
	error = hipMalloc((void**)&d_coords, memory);
	error = hipMalloc((void**)&counter, sizeof(int));
	error = hipMemcpy(d_coords, coords, memory, hipMemcpyHostToDevice);
	error = hipMemset(counter, 0, sizeof(int));

	dim3 dimblock(BLOCKSIZE);
	dim3 dimgrid((int)(ceil((double)num/3/BLOCKSIZE)));

	cu_examine<<<dimgrid, dimblock>>>(d_coords, num, counter);

	error = hipMemcpy(&answer, counter, sizeof(int), hipMemcpyDeviceToHost);

	if(error != hipSuccess)
		printf("Something went wrong!\n");	
	
	return answer;
}

__global__ void cu_examine(double* coords, int num, int *counter) 
{
	int x = 3* (blockIdx.x * blockDim.x + threadIdx.x);
	if( x >= num ) return;
	if ( (coords[x] > LOW) && (coords[x] < HIGH) && (coords[x+1] > LOW) && (coords[x+1] < HIGH) && (coords[x+2] > LOW) && (coords[x+2] < HIGH))
	{
		atomicAdd(counter,1);
	}
	
}
