#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE 1024

__global__ void cu_examine(double*,const int, int*);
double calcTime(struct timespec start,struct timespec end);

extern "C" int checkInRange(double *coords, int num)  
{
	int answer;
	int memory = num*sizeof(double);
	int * counter;
	int error = 0;
	double *d_coords;	

	error = hipMalloc((void**)&d_coords, memory);
	error = hipMalloc((void**)&counter, sizeof(int));
	error = hipMemcpy(d_coords, coords, memory, hipMemcpyHostToDevice);
	error = hipMemset(counter, 0, sizeof(int));
	struct timespec start, end;	

	dim3 dimblock(BLOCKSIZE);
	dim3 dimgrid((int)(ceil((double)num/3/BLOCKSIZE)));

	clock_gettime(CLOCK_MONOTONIC, &start);
	cu_examine<<<dimgrid, dimblock>>>(d_coords, num, counter);
	hipDeviceSynchronize();
	clock_gettime(CLOCK_MONOTONIC, &end);

	error = hipMemcpy(&answer, counter, sizeof(int), hipMemcpyDeviceToHost);

	if(error != hipSuccess)
		printf("Something went wrong!\n");
	printf("kernel exec time: %f\n",calcTime(start, end));	
		
	hipDeviceReset();
	return answer;
}

__global__ void cu_examine(double* coords, const int border, int *counter) 
{
	const int low = 12;
	const int high = 30;
	const int tx = threadIdx.x;
	const int bx = blockIdx.x;
	const int bDim = blockDim.x;
	const int b = border;

	const int x = 3* (bx * bDim + tx);
	__shared__ double s_coords[3*BLOCKSIZE];

	s_coords[tx] = coords[x];
	s_coords[tx+1] = coords[x+1];
	s_coords[tx+2] = coords[x+2];

	if( x >= b ) return;

	if ((s_coords[tx] >= low) && (s_coords[tx] <= high) && (s_coords[tx+1] >= low) && (s_coords[tx+1] <= high) && (s_coords[tx+2] >= low) && (s_coords[tx+2] <= high))
	{
		atomicAdd(counter,1);
	}	
}







double calcTime(struct timespec start, struct timespec end)
{
	const int DAS_NANO_SECONDS_IN_SEC = 1000000000;
    	long timeElapsed_s = end.tv_sec - start.tv_sec;
    	long timeElapsed_n = end.tv_nsec - start.tv_nsec;
    	if ( timeElapsed_n < 0 )
	{
        	timeElapsed_n = DAS_NANO_SECONDS_IN_SEC + timeElapsed_n;
        	timeElapsed_s--;
    	}
    	double secs = timeElapsed_s + timeElapsed_n/1000000000.0;
    	return secs;
}
